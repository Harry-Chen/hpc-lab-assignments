#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "common.h"

const char* version_name = "cuSPARSE SpTRSV";\

#define CHECK_CUSPARSE(ret) if(ret != HIPSPARSE_STATUS_SUCCESS) { fprintf(stderr, "error in line %d\n", __LINE__);}

typedef struct {
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descrA;
    csrsv2Info_t info;
    void *pBuffer;
} additional_info_t;

typedef additional_info_t *info_ptr_t;

void preprocess(dist_matrix_t *mat) {
    info_ptr_t p = (info_ptr_t)malloc(sizeof(additional_info_t));
    int pBufferSize;
    hipsparseCreate(&p->handle);
    hipsparseSetPointerMode(p->handle, HIPSPARSE_POINTER_MODE_HOST);
    hipsparseCreateMatDescr(&p->descrA);
    hipsparseSetMatFillMode(p->descrA, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(p->descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    //hipsparseSetMatIndexBase(p->descrA, HIPSPARSE_INDEX_BASE_ZERO);
    
    hipsparseCreateCsrsv2Info(&p->info);
    hipsparseDcsrsv2_bufferSize(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, mat->global_m, \
                               mat->global_nnz, p->descrA, mat->gpu_values, mat->gpu_r_pos, \
                               mat->gpu_c_idx, p->info, &pBufferSize);
    hipMalloc(&p->pBuffer, pBufferSize);
    hipsparseDcsrsv2_analysis(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, mat->global_m, \
                             mat->global_nnz, p->descrA, mat->gpu_values, mat->gpu_r_pos, \
                             mat->gpu_c_idx, p->info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, p->pBuffer);
    mat->additional_info = p;
    mat->perm = (int*) malloc(sizeof(int) * mat->global_m);
    for(int i = 0; i < mat->global_m; ++i) {
        mat->perm[i] = i;
    }
}

void destroy_additional_info(void *additional_info) {
    info_ptr_t p = (info_ptr_t)additional_info;
    hipFree(p->pBuffer);
    hipsparseDestroyCsrsv2Info(p->info);
    hipsparseDestroyMatDescr(p->descrA);
    hipsparseDestroy(p->handle);
    free(p);
}

void sptrsv(dist_matrix_t *mat, const data_t* b, data_t* x) {
    int m = mat->global_m, nnz = mat->global_nnz;
    const data_t alpha = 1.0;
    info_ptr_t p = (info_ptr_t)mat->additional_info;
    hipsparseDcsrsv2_solve(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nnz, &alpha, \
                          p->descrA, mat->gpu_values, mat->gpu_r_pos, mat->gpu_c_idx, \
                          p->info, b, x, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, p->pBuffer);
}
