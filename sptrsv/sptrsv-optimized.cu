#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include <algorithm>
#include <utility>
#include <functional>

#include "common.h"
#include "utils.h"

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 64
#endif

#ifndef AVG_NUM_THRESHOLD
#define AVG_NUM_THRESHOLD 50
#endif

#ifndef FORCE_USE_THREAD
#define FORCE_USE_THREAD false
#endif

#ifndef FORCE_USE_WARP
#define FORCE_USE_WARP true
#endif

#if FORCE_USE_THREAD && FORCE_USE_WARP
#error "cannot specify thread-only & warp-only simultaneously"
#endif

#ifndef REORDER_ROW
#define REORDER_ROW true
#endif

#ifndef SORT_COLUMN
#define SORT_COLUMN true
#endif

#ifndef DEBUG_PRINT
#define DEBUG_PRINT false
#endif

const char *version_name = "optimized version";


struct algo_info_t {
    bool use_thread = !FORCE_USE_WARP;
    bool use_warp = FORCE_USE_WARP;
    bool reorder_row = REORDER_ROW;
    bool sort_column = SORT_COLUMN;
    int block_size = BLOCK_SIZE;
};

algo_info_t select_algorithm(int m, int nnz, int level) {
    algo_info_t info;
    double avg_nnz = (double) nnz / m;
    // select thread or warp level
    info.use_thread = avg_nnz < 8 || (level >= 1250 && level < 2000);
    info.use_warp = !info.use_thread;
    // reorder_row only when using thread
    info.reorder_row = !info.use_thread;
    // select block size
    if (avg_nnz >= 50 || (avg_nnz >= 1.6 && avg_nnz < 2)) {
        info.block_size = 64;
    } else if (avg_nnz >= 5) {
        info.block_size = 128;
    } else {
        info.block_size = 256;
    }
    // whether sort columns
    info.sort_column = (avg_nnz >= 1.5 && avg_nnz <= 8) || (avg_nnz >= 9 && avg_nnz <= 10) || (avg_nnz > 10 && (
        (level >= 1250 && level < 2000) || (level >= 3000 && level < 3500) || (level >= 4000 && level < 5000)
    ));
#if DEBUG_PRINT
    printf("Selected parameters: %d %d %d %d %d\n", info.use_thread, info.use_warp, info.reorder_row, info.sort_column, info.block_size);
#endif
    return info;
}

static algo_info_t curr_algo = algo_info_t();

void preprocess(dist_matrix_t *mat) {

    int m = mat->global_m;
    int nnz = mat->global_nnz;

    int *row_offset;

    auto info = new sptrsv_info_t;
    CUDA_CHECK(hipStreamCreate(&info->copy_stream));
    CUDA_CHECK(hipMalloc(&info->c_idx_sorted, nnz * sizeof(index_t)));
    CUDA_CHECK(hipMalloc(&info->values_sorted, nnz * sizeof(data_t)));
    CUDA_CHECK(hipMalloc(&info->values_diag_inv, m * sizeof(data_t)));
    CUDA_CHECK(hipMalloc(&info->finished, m * sizeof(char)));
    CUDA_CHECK(hipMalloc(&info->curr_id, sizeof(int)));

    // pre-calculate reciprocals
    auto values_diag_inv = new data_t[m];

    auto levels = new int[m];
    levels[0] = 0;
    int max_level = 0;

    using sort_data_t = std::pair<int, std::pair<index_t, data_t>>;
    index_t *c_idx_sorted = new index_t[nnz];
    data_t *values_sorted = new data_t[nnz];
    sort_data_t *data_sort;

    memcpy(c_idx_sorted, mat->c_idx, sizeof(index_t) * nnz);
    memcpy(values_sorted, mat->values, sizeof(data_t) * nnz);

    // count max levels
    for (int i = 0; i < m; ++i) {
        int begin = mat->r_pos[i], end = mat->r_pos[i + 1];
        // level for current row
        int l = -1;
        for (int j = begin; j < end - 1; j++) {
            int col = mat->c_idx[j];
            l = max(levels[col], l);
        }
        levels[i] = l + 1;
        max_level = max(max_level, l + 1);
        values_diag_inv[i] = 1 / mat->values[end - 1]; // calcualte reciprocals on diagonal
    }

    curr_algo = select_algorithm(m, nnz, max_level + 1);

    // warp count in hybrid mode
    int k = 1;

    if (curr_algo.use_thread && curr_algo.use_warp) {
        // hybrid mode
        int curr_row = 0;
        row_offset = new int[m + 1];
        row_offset[0] = 0;
        for (int i = 0; i < m; i += 32) {
            int row_end = min(i + 32, m);
            int rows = row_end - i;
            int elements = mat->r_pos[row_end] - mat->r_pos[i];
            auto avg_per_row = (double) elements / rows;
            bool use_warp = avg_per_row >= AVG_NUM_THRESHOLD;
            if (!FORCE_USE_THREAD && (FORCE_USE_WARP || use_warp)) {
                // one warp for each row
                for (int j = 0; j < rows; ++j) {
                    row_offset[k++] = ++curr_row;
                }
            } else {
                // one thread for each row
                curr_row += rows;
                row_offset[k++] = curr_row;
            }
        }
        info->warp_count = k - 1;
        CUDA_CHECK(hipMalloc(&info->row_offset, (m + 1) * sizeof(int)));
        CUDA_CHECK(hipMemcpyAsync(info->row_offset, row_offset, k * sizeof(int), hipMemcpyHostToDevice, info->copy_stream));
    } else if (curr_algo.use_thread) {
        // thread only
        info->warp_count = ceiling(m, 32);
    } else {
        // warp only
        info->warp_count = m;
    }

    // copy sorted values to gpu asynchronously
    CUDA_CHECK(hipMemcpyAsync(info->values_diag_inv, values_diag_inv, m * sizeof(data_t), hipMemcpyHostToDevice, info->copy_stream));
    CUDA_CHECK(hipMemcpyAsync(info->c_idx_sorted, c_idx_sorted, nnz * sizeof(index_t), hipMemcpyHostToDevice, info->copy_stream));
    CUDA_CHECK(hipMemcpyAsync(info->values_sorted, values_sorted, nnz * sizeof(data_t), hipMemcpyHostToDevice, info->copy_stream));

    // sort data on each row according to levels
    if (curr_algo.sort_column) {
        data_sort = new sort_data_t[m];
        for (int i = 0; i < m; ++i) {
            int begin = mat->r_pos[i], end = mat->r_pos[i + 1];
            for (int j = begin; j < end - 1; j++) {
                int col = mat->c_idx[j];
                data_sort[j - begin] = std::make_pair(levels[col], std::make_pair(col, mat->values[j]));
            }
            if (end > begin + 1) {
                std::sort(data_sort, data_sort + end - begin - 1, [&](const sort_data_t &i, const sort_data_t &j) { return i.first < j.first; });
                for (int j = begin; j < end - 1; j++) {
                    const auto &data = data_sort[j - begin].second;
                    c_idx_sorted[j] = data.first;
                    values_sorted[j] = data.second;
                }
            }
        }
    }

    // count number of each levels then reorder according to levels
    if (curr_algo.reorder_row) {
        CUDA_CHECK(hipMalloc(&info->row_orders, m * sizeof(index_t)));

        auto level_offsets = new int[max_level + 2](), level_counts = new int[max_level + 1]();
        auto row_orders = new index_t[m];

        // counting sort
        for (int i = 0; i < m; ++i) {
            level_offsets[levels[i] + 1]++;
        }
        for (int i = 0; i < max_level + 1; ++i) {
            level_offsets[i + 1] += level_offsets[i];
        }
        for (int i = 0; i < m; ++i) {
            int level = levels[i];
            int new_order = level_offsets[level] + (level_counts[level]++);
            row_orders[new_order] = i;
        }

        // copy new row orders to GPU
        CUDA_CHECK(hipMemcpyAsync(info->row_orders, row_orders, m * sizeof(index_t), hipMemcpyHostToDevice, info->copy_stream));
    }

    mat->additional_info = info;
}

void destroy_additional_info(void *additional_info) {
    hipStreamDestroy(((sptrsv_info_t *)additional_info)->copy_stream);
}

__global__ void sptrsv_capellini_warp_kernel(
    const index_t *__restrict__ r_pos, const index_t *__restrict__ c_idx, const data_t *__restrict__ values, const data_t *__restrict__ values_diag_inv, const index_t *__restrict__ row_orders,
    const int *__restrict__ row_offset, const int warp_count, const int m, const data_t *__restrict__ b, data_t *__restrict__ x, volatile char *__restrict__ finished, int *curr_id, bool reorder_row
) {

    const int lane_id = threadIdx.x & 31;

    // allocate thread id according to scheduling order
    int id = 0;
    if (lane_id == 0) {
        id = atomicAdd(curr_id, 1);
    }
    id = __shfl_sync(0xFFFFFFFF, id, 0);

    // row id
    int i = id;

    while (i < m) {
        // assign one warp for current row
        if (reorder_row) i = row_orders[i];

        data_t left_sum = 0;
        const int begin = r_pos[i], end = r_pos[i + 1];
        data_t bi = b[i], diag_inv = values_diag_inv[i];
        bi *= diag_inv;
            
        // calculate sum of previous columns
#pragma unroll
	for (int j = begin + lane_id; j < end - 1; j += 32) {
            data_t value = values[j];
            int col = c_idx[j];
            while (finished[col] == 0) {
                __threadfence();
            }
            __threadfence();
            left_sum += value * x[col];
        }

        left_sum *= diag_inv;
    
        // reduce within warp
#pragma unroll
        for (int offset = 16; offset > 0; offset >>= 1) {
            left_sum += __shfl_down_sync(0xFFFFFFFF, left_sum, offset);
        }
    
        // write back final results
        if (lane_id == 0) {
            x[i] = bi - left_sum;
            __threadfence();
            finished[i] = 1;
            id = atomicAdd(curr_id, 1);
        }

        id = __shfl_sync(0xFFFFFFFF, id, 0);
        i = id;
    }
}


template <bool FORCE_THREAD = FORCE_USE_THREAD, bool FORCE_WARP = FORCE_USE_WARP>
__global__ void sptrsv_capellini_adaptive_kernel(
    const index_t *__restrict__ r_pos, const index_t *__restrict__ c_idx, const data_t *__restrict__ values, const data_t *__restrict__ values_diag_inv, const index_t *__restrict__ row_orders,
    const int *__restrict__ row_offset, const int warp_count, const int m, const data_t *__restrict__ b, data_t *__restrict__ x, volatile char *__restrict__ finished, int *curr_id, bool reorder_row
) {

    bool need_continue = true;
    const int lane_id = threadIdx.x & 31;

    do {
        // allocate thread id according to scheduling order
        int id = 0;
        if (lane_id == 0) {
            id = atomicAdd(curr_id, 1) << 5;
        }
        id = __shfl_sync(0xFFFFFFFF, id, 0) + lane_id;
        const int w = id >> 5;
        
        if (!FORCE_THREAD && w >= warp_count) {
            return;
        } 

        // whether use thread (thread-only or hybrid mode)
        bool use_thread = FORCE_THREAD || (!FORCE_WARP && row_offset[w + 1] > row_offset[w] + 1);

        // row id
        int i;
        if (FORCE_THREAD) {
            i = id;
        } else if (FORCE_WARP) {
            i = w;
        } else if (use_thread) {
            i = row_offset[w] + lane_id;
        } else {
            i = row_offset[w];
        }

        if (FORCE_THREAD || (!FORCE_WARP && use_thread)) {
            // assign one thread for current row

            if (i >= m) {
                need_continue = false;
                break;
            }
            if (reorder_row) i = row_orders[i];

            const int begin = r_pos[i], end = r_pos[i + 1];
            data_t bi = b[i], diag_inv = values_diag_inv[i];
            
            for (int j = begin; j < end;) {
                int col = c_idx[j];
                // write back results
                if (col == i) {
                    x[i] = bi * diag_inv;
                    __threadfence();
                    finished[col] = 1;
                    break;
                }
                // wait for col to finish
                while (finished[col]) {
                    bi -= values[j] * x[col];
                    col = c_idx[++j];
                }
            }
        } else {
            // assign one warp for current row

            if (i >= m) {
                need_continue = false;
                break;
            }
            if (reorder_row) i = row_orders[i];

            data_t left_sum = 0;
            const int begin = r_pos[i], end = r_pos[i + 1];
            data_t bi = b[i], diag_inv = values_diag_inv[i];
            bi *= diag_inv;
                
            // calculate sum of previous columns
            for (int j = begin + lane_id; j < end - 1; j += 32) {
                data_t value = values[j];
                int col = c_idx[j];
                while (finished[col] == 0) {
                    __threadfence();
                }
                __threadfence();
                left_sum += value * x[col];
            }

            left_sum *= diag_inv;
        
            // reduce within warp
            for (int offset = 16; offset > 0; offset >>= 1) {
                left_sum += __shfl_down_sync(0xFFFFFFFF, left_sum, offset);
            }
        
            // write back final results
            if (lane_id == 0) {
                x[i] = bi - left_sum;
                __threadfence();
                finished[i] = 1;
            }
        }

    } while (need_continue);
}


void sptrsv(dist_matrix_t *mat, const data_t *__restrict__ b, data_t *__restrict__ x) {
    int m = mat->global_m;

    auto info = (sptrsv_info_t *) mat->additional_info;
    auto finished = info->finished;
    auto curr_id = info->curr_id;

    // clear flags
    CUDA_CHECK(hipMemset(finished, 0, m * sizeof(char)));
    CUDA_CHECK(hipMemset(curr_id, 0, sizeof(int)));

    int block_size = curr_algo.block_size;

    // select algorithms with different template types
    if (curr_algo.use_thread && !curr_algo.use_warp) {
        // thread only
        assert(false);
        sptrsv_capellini_adaptive_kernel<true, false><<<ceiling(m, block_size), block_size>>>(mat->gpu_r_pos, info->c_idx_sorted, info->values_sorted, info->values_diag_inv, info->row_orders, info->row_offset, info->warp_count, m, b, x, finished, curr_id, curr_algo.reorder_row);
    } else if (!curr_algo.use_thread && curr_algo.use_warp) {
        // // warp only
        // sptrsv_capellini_warp_kernel<<<90, 64>>>(mat->gpu_r_pos, info->c_idx_sorted, info->values_sorted, info->values_diag_inv, info->row_orders, info->row_offset, info->warp_count, m, b, x, finished, curr_id, curr_algo.reorder_row);
        sptrsv_capellini_adaptive_kernel<false, true><<<90, 64>>>(mat->gpu_r_pos, info->c_idx_sorted, info->values_sorted, info->values_diag_inv, info->row_orders, info->row_offset, info->warp_count, m, b, x, finished, curr_id, curr_algo.reorder_row);
    } else {
        // hybrid mode
        assert(false);
        sptrsv_capellini_adaptive_kernel<false, false><<<ceiling(m * 32, block_size), block_size>>>(mat->gpu_r_pos, info->c_idx_sorted, info->values_sorted, info->values_diag_inv, info->row_orders, info->row_offset, info->warp_count, m, b, x, finished, curr_id, curr_algo.reorder_row);
    }

    CUDA_CHECK(hipGetLastError());
}
