#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "common.h"
#include "utils.h"

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024
#endif

#ifndef AVG_NUM_THRESHOLD
#define AVG_NUM_THRESHOLD 10
#endif

const char* version_name = "optimized version";

void preprocess(dist_matrix_t *mat) {

    int m = mat->global_m;
    int nnz = mat->global_nnz;
    int curr_row = 0;

    auto row_offset = new int[m + 1];

    row_offset[0] = 0;
    int k = 1;

    for (int i = 0; i < m; i += 32) {
        int row_end = min(i + 32, m);
        int rows = row_end - i;
        int elements = mat->r_pos[row_end] - mat->r_pos[i];
        auto avg_per_row = (double) elements / rows;
        bool use_warp = avg_per_row >= AVG_NUM_THRESHOLD;
        if (use_warp) {
            // one warp for each row
            for (int j = 0; j < rows; ++j) {
                row_offset[k++] = ++curr_row;
            }
        } else {
            // one thread for each row
            curr_row += rows;
            row_offset[k++] = curr_row;
        }
    }

    auto info = new sptrsv_info_t;
    info->warp_count = k - 1;
    CUDA_CHECK(hipMalloc(&info->finished, m * sizeof(int)));
    CUDA_CHECK(hipMalloc(&info->row_offset, (m + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&info->curr_id, sizeof(int)));
    CUDA_CHECK(hipMemcpy(info->row_offset, row_offset, k * sizeof(int), hipMemcpyHostToDevice));
    mat->additional_info = info;

    delete[] row_offset;
}

void destroy_additional_info(void *additional_info) {}


__global__ void sptrsv_capellini_thread_kernel(
    const index_t *__restrict__ r_pos, const index_t *__restrict__ c_idx, const data_t *__restrict__ values,
    const int m, const int nnz, const data_t *__restrict__ b, data_t *__restrict__ x, int *finished, int *curr_id
) {
    
    // allocate thread id by scheduling order
    const int i = atomicAdd(curr_id, 1);
    if (i >= m) return;


    // begin index of current warp (must be contiguous 32 numbers in a row)
    const int warp_begin = (i >> 5) << 5;

    data_t left_sum = 0;

    const int begin = r_pos[i], end = r_pos[i + 1];

    int j = begin;

    // for (; j < end; ++j) {
    //     int col = c_idx[j];
    //     if (col < warp_begin) {
    //         while (finished[col] != 1) __threadfence_block();
    //         left_sum += values[j] * x[col];
    //     } else {
    //         break;
    //     }
    // }

    // go through all numbers on current row
    while (j < end) {
    // for (int k = 0; k < 16; ++k) {
        int col = c_idx[j];
        // iterate over all finished numbers
        volatile int *finished_col = finished + col;
        while (finished[col] != 1) {}
            left_sum += values[j] * x[col];
            col = c_idx[++j];
        // }
        // last number (on diagonal)
        if (col == i) {
            x[i] = (b[i] - left_sum) / values[end - 1];
            __threadfence(); // ensure x[i] can be read properly by other threads
            finished[i] = 1;
            ++j;
        }
    }
}


__global__ void sptrsv_capellini_warp_kernel(
    const index_t *__restrict__ r_pos, const index_t *__restrict__ c_idx, const data_t *__restrict__ values, const int *__restrict__ row_offset,
    const int warp_count, const int m, const int nnz, const data_t *__restrict__ b, data_t *__restrict__ x, volatile int *finished, int *curr_id
) {
    
    // allocate thread id by scheduling order
    const int id = atomicAdd(curr_id, 1);
    const int w = id >> 5;
    const int lane_id = id & 31;
    if (w >= warp_count) return;

    bool use_thread = row_offset[w + 1] > row_offset[w] + 1;

    if (use_thread) {
        // one thread for current row
        int i = row_offset[w] + lane_id;
        if (i >= m) return;

        data_t left_sum = 0;
        const int begin = r_pos[i], end = r_pos[i + 1];

        int j = begin;
        while (j < end) {
            int col = c_idx[j];
            while (finished[col] == 1) {
                left_sum += values[j] * x[col];
                col = c_idx[++j];
            }
            if (col == i) {
                x[i] = (b[i] - left_sum) / values[end - 1];
                __threadfence();
                finished[i] = 1;
                // atomicAdd(&finished[i], 1);
                ++j;
            }
        }
    } else {
        // one warp for current row
        int i = row_offset[w];
        if (i >= m) return;

        data_t left_sum = 0;
        const int begin = r_pos[i], end = r_pos[i + 1];
    
        // calculate sum of previous columns
        for (int j = begin + lane_id; j < end - 1;) {
            int col = c_idx[j];
            while (finished[col] != 1) {}
            left_sum += values[j] * x[col];
            j += 32;
        }
    
        // reduce within warp
        for (int offset = 16; offset > 0; offset >>= 1) {
            left_sum += __shfl_down_sync(0xFFFFFFFF, left_sum, offset);
        }
    
        if (lane_id == 0) {
            x[i] = (b[i] - left_sum) / values[end - 1];
            __threadfence();
            finished[i] = 1;
            // atomicAdd(&finished[i], 1);
        }
    }
}


void sptrsv(dist_matrix_t *mat, const data_t *__restrict__ b, data_t *__restrict__ x) {
    int m = mat->global_m;
    int nnz = mat->global_nnz;

    auto info = (sptrsv_info_t *) mat->additional_info;
    auto finished = info->finished;
    auto curr_id = info->curr_id;
    CUDA_CHECK(hipMemset(finished, 0, m * sizeof(int)));
    CUDA_CHECK(hipMemset(curr_id, 0, sizeof(int)));

    // sptrsv_capellini_thread_kernel<<<ceiling(m, BLOCK_SIZE), BLOCK_SIZE>>>(mat->gpu_r_pos, mat->gpu_c_idx, mat->gpu_values, m, nnz, b, x, finished, curr_row);
    sptrsv_capellini_warp_kernel<<<ceiling(m * 32, BLOCK_SIZE), BLOCK_SIZE>>>(mat->gpu_r_pos, mat->gpu_c_idx, mat->gpu_values, info->row_offset, info->warp_count, m, nnz, b, x, finished, curr_id);
    CUDA_CHECK(hipGetLastError());
}
