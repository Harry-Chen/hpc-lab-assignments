#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "utils.h"

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024
#endif

const char* version_name = "optimized version";

void preprocess(dist_matrix_t *mat) {
    auto info = new sptrsv_info_t;
    CUDA_CHECK(hipMalloc(&info->finished, mat->global_m * sizeof(int)));
    CUDA_CHECK(hipMalloc(&info->curr_row, sizeof(int)));
    mat->additional_info = info;
}

void destroy_additional_info(void *additional_info) {}


__global__ void sptrsv_capellini_thread_kernel(
    const index_t *__restrict__ r_pos, const index_t *__restrict__ c_idx, const data_t *__restrict__ values,
    const int m, const int nnz, const data_t *__restrict__ b, data_t *__restrict__ x, int *finished, int *curr_row
) {
    
    // allocate thread id by scheduling order
    const int i = atomicAdd(curr_row, 1);
    if (i >= m) return;


    // begin index of current warp (must be contiguous 32 numbers in a row)
    const int warp_begin = (i >> 5) << 5;

    data_t left_sum = 0;

    const int begin = r_pos[i], end = r_pos[i + 1];

    int j = begin;

    // for (; j < end; ++j) {
    //     int col = c_idx[j];
    //     if (col < warp_begin) {
    //         while (finished[col] != 1) __threadfence_block();
    //         left_sum += values[j] * x[col];
    //     } else {
    //         break;
    //     }
    // }

    // go through all numbers on current row
    while (j < end) {
    // for (int k = 0; k < 16; ++k) {
        int col = c_idx[j];
        // iterate over all finished numbers
        volatile int *finished_col = finished + col;
        while (finished[col] == 1) {
            left_sum += values[j] * x[col];
            col = c_idx[++j];
        }
        // last number (on diagonal)
        if (col == i) {
            x[i] = (b[i] - left_sum) / values[end - 1];
            __threadfence(); // ensure x[i] can be read properly by other threads
            finished[i] = 1;
            ++j;
        }
    }
}


__global__ void sptrsv_capellini_warp_kernel(
    const index_t *__restrict__ r_pos, const index_t *__restrict__ c_idx, const data_t *__restrict__ values,
    const int m, const int nnz, const data_t *__restrict__ b, data_t *__restrict__ x, int *finished, int *curr_row
) {
    
    // allocate thread id by scheduling order
    const int i = atomicAdd(curr_row, 1);
    const int w = i >> 5;
    const int lane_id = i & 31;
    if (w >= m) return;

    data_t left_sum = 0;

    const int begin = r_pos[w], end = r_pos[w + 1];

    // calculate sum of previous columns
    for (int j = begin + lane_id; j < end - 1;) {
        int col = c_idx[j];
        volatile int *finished_col = finished + col;
        while (*finished_col == 0);
        left_sum += values[j] * x[col];
        j += 32;
    }

    // reduce within warp
    for (int offset = 16; offset > 0; offset >>= 1) {
        left_sum += __shfl_down_sync(0xFFFFFFFF, left_sum, offset);
    }

    if (lane_id == 0) {
        x[w] = (b[w] - left_sum) / values[end - 1];
        __threadfence();
        finished[w] = 1;
    }
}


void sptrsv(dist_matrix_t *mat, const data_t *__restrict__ b, data_t *__restrict__ x) {
    int m = mat->global_m;
    int nnz = mat->global_nnz;

    auto info = (sptrsv_info_t *) mat->additional_info;
    auto finished = info->finished;
    auto curr_row = info->curr_row;
    CUDA_CHECK(hipMemset(finished, 0, m * sizeof(int)));
    CUDA_CHECK(hipMemset(curr_row, 0, sizeof(int)));

    // sptrsv_capellini_thread_kernel<<<ceiling(m, BLOCK_SIZE), BLOCK_SIZE>>>(mat->gpu_r_pos, mat->gpu_c_idx, mat->gpu_values, m, nnz, b, x, finished, curr_row);
    sptrsv_capellini_warp_kernel<<<ceiling(m * 32, BLOCK_SIZE), BLOCK_SIZE>>>(mat->gpu_r_pos, mat->gpu_c_idx, mat->gpu_values, m, nnz, b, x, finished, curr_row);
    CUDA_CHECK(hipGetLastError());
}
