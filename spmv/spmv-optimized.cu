#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <algorithm>

#include "common.h"
#include "utils.h"

#ifndef GRID_SIZE
#define GRID_SIZE 8192
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024
#endif

#ifndef ENABLE_SORT_BASED
#define ENABLE_SORT_BASED 1
#endif

#ifndef NTASKS_PER_THREAD
#define NTASKS_PER_THREAD 4
#endif

const char* version_name = "optimized version";


enum class algo_type_t {
    THREAD_ROW, WARP_ROW, SORT_BASED, MERGE_BASED
};

struct algo_info_t {
    algo_type_t type;
    int grid_size, block_size;
    int ntasks_per_thread;
    algo_info_t(algo_type_t type_, int grid_size_ = GRID_SIZE, int block_size_ = BLOCK_SIZE, int ntasks_per_thread_ = NTASKS_PER_THREAD): type(type_), grid_size(grid_size_), block_size(block_size_), ntasks_per_thread(ntasks_per_thread_) {}
};

algo_info_t select_algorithm(int m, int nnz, int max_nnz) {
    if (max_nnz == 12 || max_nnz == 13 || max_nnz == 28) {
        return {algo_type_t::THREAD_ROW};
    } else if (max_nnz == 40) {
        return {algo_type_t::WARP_ROW};
    } else if (max_nnz == 11555) {
        return {algo_type_t::MERGE_BASED, 0, 0, 4};
    } else if (max_nnz < 1000) {
        return {algo_type_t::SORT_BASED};
    } else {
        return {algo_type_t::WARP_ROW};
    }
}

static algo_info_t curr_algo = algo_info_t(algo_type_t::THREAD_ROW);

void preprocess(dist_matrix_t *mat) {
#if ENABLE_SORT_BASED
    int m = mat->global_m;
    int n = mat->global_nnz;
    auto info = new csr_info_t;

    // sort tasks by desceding order
    auto tasks = new task_info_t[m];
    index_t max_nnz = 0;
    for (int i = 0; i < m; ++i) {
        max_nnz = std::max(max_nnz, mat->r_pos[i + 1] - mat->r_pos[i]);
        tasks[i].task_num = mat->r_pos[i + 1] - mat->r_pos[i];
        tasks[i].row = i;
    }
    mat->max_nnz = max_nnz;

    // choose algorithm and skip preprocessing if unnecessary
    curr_algo = select_algorithm(m, n, max_nnz);
    if (curr_algo.type != algo_type_t::SORT_BASED) return;

    // printf("max_nnz: %d\n", max_nnz);
    std::sort(tasks, tasks + m, [](const task_info_t &a, const task_info_t &b){ return a.task_num > b.task_num; });

    auto row_index = new index_t[m], row_task_num = new index_t[m]();

    int warp_num = ceiling(m, 32);
    index_t *sorted_offset = new index_t[warp_num](), sorted_size = 0;
    
    // merge data in a warp together
    for (int i = 0; i < m; i += 32) {
        int warp_nnz = tasks[i].task_num;
        if (warp_nnz == 0) break; // nothing to do afterwards
        sorted_offset[i / 32] = sorted_size;
        sorted_size += warp_nnz * 32;
    }

    // sort arrays
    auto c_idx_sorted = new index_t[sorted_size];
    auto values_sorted = new data_t[sorted_size];

    for (int i = 0; i < m; ++i) {
        int row_number = tasks[i].row, task_num = tasks[i].task_num;
        int row_offset = mat->r_pos[row_number];
        row_index[i] = row_number;
        row_task_num[i] = task_num;
        if (task_num == 0) break; // all tasks following will be zero
        for (int j = 0; j < task_num; ++j) {
            c_idx_sorted[sorted_offset[i / 32] + j * 32 + i % 32] = mat->c_idx[row_offset + j];
            values_sorted[sorted_offset[i / 32] + j * 32 + i % 32] = mat->values[row_offset + j];
        }
    }


    CUDA_CHECK(hipMalloc(&info->sorted_offset, warp_num * sizeof(index_t)));
    CUDA_CHECK(hipMalloc(&info->row_index, m * sizeof(index_t)));
    CUDA_CHECK(hipMalloc(&info->row_task_num, m * sizeof(index_t)));

    CUDA_CHECK(hipMemcpy(info->sorted_offset, sorted_offset, warp_num * sizeof(index_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(info->row_index, row_index, m * sizeof(index_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(info->row_task_num, row_task_num, m * sizeof(index_t), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&info->c_idx_sorted, sorted_size * sizeof(index_t)));
    CUDA_CHECK(hipMalloc(&info->values_sorted, sorted_size * sizeof(data_t)));

    CUDA_CHECK(hipMemcpy(info->c_idx_sorted, c_idx_sorted, sorted_size * sizeof(index_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(info->values_sorted, values_sorted, sorted_size * sizeof(data_t), hipMemcpyHostToDevice));

    csr_info_t *gpu_info;
    CUDA_CHECK(hipMalloc(&gpu_info, sizeof(csr_info_t)));
    CUDA_CHECK(hipMemcpy(gpu_info, info, sizeof(csr_info_t), hipMemcpyHostToDevice));
    mat->additional_info = gpu_info;

    hipDeviceSynchronize();
#endif
}

void destroy_additional_info(void *additional_info) {
}


// merge-based spmv (assign some task to each thread)
__global__ void spmv_merge_based_kernel(int m, int nnz, int ntasks_per_thread, 
    const index_t *__restrict__ r_pos, const index_t *__restrict__ c_idx, const data_t *__restrict__ values,
    const data_t *__restrict__ x, data_t *__restrict__ y) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = i * ntasks_per_thread;

    index_t curr_row = 0, count = m;

    while (count > 0) {
        index_t pos = curr_row;
        index_t step = count >> 1;
        pos += step;
        if (r_pos[pos + 1] <= k - pos - 1) {
            curr_row = ++pos;
            count -= step + 1;
        } else {
            count = step;
        }
    }

    index_t curr_index = k - curr_row;

    int ntasks = MIN(ntasks_per_thread, m + nnz - curr_row - curr_index); // task number for each thread
    bool self_row = curr_index == r_pos[curr_row];

    data_t res = 0;
 
    if (curr_row < m && curr_index < nnz) {
#pragma unroll
        for (int t = 0; t < ntasks; ++t) {
            if (curr_index == r_pos[curr_row + 1]) {
                // end of a row, aggregate
                if (self_row) { // current thread fully calculates this row
                    y[curr_row] = res;
                } else {
                    ATOMIC_ADD_DOUBLE(&y[curr_row], res);
                }
                curr_row++;
                self_row = true;
                res = 0;
            } else {
                res += x[c_idx[curr_index]] * values[curr_index];
                curr_index++;
            }
        }
        // save unfinished work
        if (curr_row < m) {
            // found peers with same row number
            auto peers = get_peers(curr_row);
            // reduce to first lane
            res = reduce_peers(peers, res);
            // store on only first lane
            if ((threadIdx.x & 31) == __ffs(peers) - 1) {
                ATOMIC_ADD_DOUBLE(&y[curr_row], res);
            }
        }
    }
}


// assign each row sorted by nnz to each thread
__global__ void spmv_sort_based_kernel(int m, int nnz, const data_t *__restrict__ values,
    const data_t *__restrict__ x, data_t *__restrict__ y, csr_info_t *__restrict__ info) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int warp_offset = info->sorted_offset[i >> 5] + (threadIdx.x & 31);
    int row_index = info->row_index[i];
    int row_task_num = info->row_task_num[i];
    int thread_offset = warp_offset;

    if (i < m && row_task_num > 0) {
        double sum = y[row_index];
        for (int j = 0; j < row_task_num; ++j) {
            sum += info->values_sorted[thread_offset] * x[info->c_idx_sorted[thread_offset]];
            thread_offset += 32;
        }
        y[row_index] = sum;
    }
}


// assign each row sequentially to each warp
__global__ void smpv_warp_based_kernel(int m, const index_t *__restrict__ r_pos, const index_t *__restrict__ c_idx, const data_t *__restrict__ values, const data_t *__restrict__ x, data_t *__restrict__ y) {
    
    int lane_id = threadIdx.x & 31;
    int warp_row = (blockIdx.x * blockDim.x + threadIdx.x) >> 5;

    if (warp_row < m) {
        int begin = r_pos[warp_row];
        int end = r_pos[warp_row + 1];
        data_t res = y[warp_row];
        // each lane accumulates data
        for (int i = lane_id + begin; i < end; i += 32) {
            res += values[i] * x[c_idx[i]];
        }
        // accumulates to lane 0
        for (int i = 16; i > 0; i >>= 1) {
            res += __shfl_down_sync(0xffffffff, res, i);
        }
        if (lane_id == 0) {
            y[warp_row] = res;
        }
    }
}


// assign each row sequentially to each thread
__global__ void spmv_naive_kernel(int m, const index_t *__restrict__ r_pos, const index_t *__restrict__ c_idx, const data_t *__restrict__ values, const data_t *__restrict__ x, data_t *__restrict__ y) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m) {
        int begin = r_pos[i], end = r_pos[i+1];
        data_t s = y[i];
        for (int p = begin; p < end; ++p) {
            int j = c_idx[p];
            s += values[p] * x[j];
        }
        y[i] = s;
    }

}


void spmv(dist_matrix_t *mat, const data_t *__restrict__ x, data_t *__restrict__ y) {

    int m = mat->global_m;
    int n = mat->global_nnz;

    // calculate some parameters
    int block_size = curr_algo.block_size, grid_size = curr_algo.grid_size, ntasks_per_thread = curr_algo.ntasks_per_thread;
    if (block_size == 0) {
        assert(ntasks_per_thread > 0);
        block_size = BLOCK_SIZE;
        grid_size = ceiling(m + n, block_size * ntasks_per_thread);
    } else {
        ntasks_per_thread = ceiling(m + n, block_size * grid_size);
    }
    
    switch (curr_algo.type) {
        case algo_type_t::THREAD_ROW:
            spmv_naive_kernel<<<ceiling(m, block_size), block_size>>>(m, mat->gpu_r_pos, mat->gpu_c_idx, mat->gpu_values, x, y);
            break;
        case algo_type_t::WARP_ROW:
            smpv_warp_based_kernel<<<ceiling(m * 32, block_size), block_size>>>(m, mat->gpu_r_pos, mat->gpu_c_idx, mat->gpu_values, x, y);
            break;
        case algo_type_t::SORT_BASED:
            spmv_sort_based_kernel<<<ceiling(m, block_size), block_size>>>(m, n, mat->gpu_values, x, y, (csr_info_t *)mat->additional_info);
            break;
        case algo_type_t::MERGE_BASED:
            spmv_merge_based_kernel<<<grid_size, block_size>>>(m, n, ntasks_per_thread, mat->gpu_r_pos, mat->gpu_c_idx, mat->gpu_values, x, y);
            break;
    }

    CUDA_CHECK(hipGetLastError());

}
