#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "common.h"

const char* version_name = "cuSPARSE SpMV";\

#define CHECK_CUSPARSE(ret) if(ret != HIPSPARSE_STATUS_SUCCESS) { fprintf(stderr, "error in line %d\n", __LINE__);}

typedef struct {
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descrA;
} additional_info_t;

typedef additional_info_t *info_ptr_t;

void preprocess(dist_matrix_t *mat) {
    info_ptr_t p = (info_ptr_t)malloc(sizeof(additional_info_t));
    hipsparseCreate(&p->handle);
    hipsparseSetPointerMode(p->handle, HIPSPARSE_POINTER_MODE_HOST);
    hipsparseCreateMatDescr(&p->descrA);
    //hipsparseSetMatIndexBase(p->descrA, HIPSPARSE_INDEX_BASE_ZERO);
    //hipsparseSetMatType(p->descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    mat->additional_info = p;
}

void destroy_additional_info(void *additional_info) {
    info_ptr_t p = (info_ptr_t)additional_info;
    hipsparseDestroy(p->handle);
    hipsparseDestroyMatDescr(p->descrA);
    free(p);
}

void spmv(dist_matrix_t *mat, const data_t* x, data_t* y) {
    int m = mat->global_m, nnz = mat->global_nnz;
    const data_t alpha = 1.0, beta = 1.0;
    info_ptr_t p = (info_ptr_t)mat->additional_info;

    hipsparseDcsrmv(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, m, nnz, 
        &alpha, p->descrA, mat->gpu_values, mat->gpu_r_pos, mat->gpu_c_idx, x, &beta, y);
}
